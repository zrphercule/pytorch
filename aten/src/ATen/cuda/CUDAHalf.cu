#include "hip/hip_runtime.h"
#include "ATen/core/Half.h"
#include "ATen/cuda/CUDAHalf.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

namespace at {
#if CUDA_VERSION < 9000 && !defined(__HIP_PLATFORM_HCC__)
template <> AT_CUDA_API
half convert(Half aten_half) {
  return half{aten_half.x};
}

template <> AT_CUDA_API
half convert(double value) {
  return half{Half(value).x};
}

template <> AT_CUDA_API
Half convert(half cuda_half) {
  return Half(cuda_half.x, Half::from_bits);
}
#else
template <> AT_CUDA_API
half convert(Half aten_half) {
  __half_raw x_raw;
  x_raw.x = aten_half.x;
  return half(x_raw);
}

template <> AT_CUDA_API
Half convert(half cuda_half) {
  __half_raw raw(cuda_half);
  return Half(raw.x, Half::from_bits);
}

template <> AT_CUDA_API
half convert(double value) {
  __half_raw raw;
  raw.x = Half(value).x;
  return half {raw};
}

template <> __half HalfFix(Half h) {
  __half_raw raw;
  raw.x = h.x;
  return __half{raw};
}

template <> Half HalfFix(__half h) {
  __half_raw raw(h);
  return Half(raw.x, Half::from_bits);
}
#endif
} // namespace at
