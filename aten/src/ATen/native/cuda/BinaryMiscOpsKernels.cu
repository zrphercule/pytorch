#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/BinaryOps.h>


// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void atan2_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "atan2_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return ::atan2(a, b);
    });
  });
}

void logical_xor_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, iter.common_dtype(), "logical_xor_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return static_cast<scalar_t>(bool(a) != bool(b));
    });
  });
}

void smooth_l1_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND(kHalf, iter.dtype(), "smooth_l1_cuda", [&]() {
    gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      auto z = fabs(a - b);
      return z < scalar_t(1.) ? scalar_t(0.5) * z * z : z - scalar_t(0.5);
    });
  });
}

void mse_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "mse_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      auto diff = a - b;
      return diff * diff;
    });
  });
}

REGISTER_DISPATCH(atan2_stub, &atan2_kernel_cuda);
REGISTER_DISPATCH(logical_xor_stub, &logical_xor_kernel_cuda);
REGISTER_DISPATCH(smooth_l1_stub, &smooth_l1_kernel_cuda);
REGISTER_DISPATCH(mse_stub, &mse_kernel_cuda);

}} // namespace at::native
