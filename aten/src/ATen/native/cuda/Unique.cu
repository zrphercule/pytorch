#include "hip/hip_runtime.h"
#include "ATen/ATen.h"
#include "ATen/cuda/HIPContext.h"
#include <THC/THCGeneral.h>
#include <THC/THCThrustAllocator.cuh>
#include <thrust/execution_policy.h>

#include <tuple>
#include <thrust/unique.h>
#include <thrust/sort.h>

namespace at {
namespace native{

#ifndef __HIP_PLATFORM_HCC__

namespace {
template <typename scalar_t>
__global__ void inverse_indices_kernel(
    const scalar_t* input_data,
    const scalar_t* output_data,
    int64_t* inverse_indices_data,
    int64_t num_inp,
    int64_t num_out) {
    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t stride = blockDim.x * gridDim.x;

    for (int64_t i = idx; i < num_inp * num_out; i += stride) {
      if (input_data[i / num_out] == output_data[i % num_out]){
        inverse_indices_data[i / num_out] = i % num_out;   
      }
    }
  }


template <typename scalar_t>
  std::tuple<Tensor, Tensor> _unique_cuda_template(
    const Tensor& self,
    const bool return_inverse) {

    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
    auto policy = thrust::cuda::par(allocator).on(stream);

    const Tensor& input = self.contiguous();
    int64_t num_inp = input.numel();
    const scalar_t* input_data = input.data<scalar_t>();

    //sort & unique
    Tensor output = input.clone();
    output = output.view(-1);
    scalar_t* output_data = output.data<scalar_t>();
    thrust::sort(policy, output_data, output_data + num_inp);
    scalar_t* output_end = thrust::unique(policy, output_data, output_data + num_inp);
    int64_t num_out = output_end - output_data;
    output.resize_(num_out);

    Tensor inverse_indices = at::empty({0}, self.type().toScalarType(kLong));

    if (return_inverse) {
      inverse_indices.resize_(input.sizes());
      int64_t* inverse_indices_data = inverse_indices.data<int64_t>();
      int block = 512;
      int grid = std::min<int64_t>((num_inp * num_out + block - 1) / block, 2048L);
      inverse_indices_kernel<<<grid, block, 0, stream>>>(
        input_data, output_data, inverse_indices_data, num_inp, num_out);
    }

    THCudaCheck(hipGetLastError());   
    return std::tuple<Tensor, Tensor>(output, inverse_indices);

  }
} // namespace

#endif

std::tuple<Tensor, Tensor>
_unique_cuda(const Tensor& self, const bool sorted, const bool return_inverse) {
#ifndef __HIP_PLATFORM_HCC__
  return AT_DISPATCH_ALL_TYPES(self.type(), "unique", [&] {
    // The current CUDA implementation of unique always sort due to the
    // lack of hashtable implementation in thrust
    return _unique_cuda_template<scalar_t>(self, return_inverse);
  });
#else
  AT_ERROR("unique_cuda: HIP not supported");
#endif
}

}  // namespace native
}  // namespace at
