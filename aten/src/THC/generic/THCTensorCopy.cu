
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorCopy.cu"
#else

THC_API void
THCTensor_(copy)(THCState* state, THCTensor* dst, THCTensor* src) {
  if (dst == src) return;
  THC_copyTensor<real, real>(state, dst, src);
}

template <>
THCTensor *THCTensor_newClone<real>(THCState *state, THCTensor *self) {
  THCTensor *tensor = THCTensor_new(state, THTensor_getStoragePtr(self)->scalar_type);
  THCTensor_resizeAs(state, tensor, self);
  THC_copyTensor<real, real>(state, tensor, self);
  return tensor;
}

template <>
THCTensor *THCTensor_newContiguous<real>(THCState *state, THCTensor *self)
{
  if(!THCTensor_isContiguous(state, self)) {
    return THCTensor_newClone<real>(state, self);
  } else {
    THCTensor_retain(state, self);
    return self;
  }
}


template <>
void THCTensor_freeCopyTo<real>(THCState *state, THCTensor *self, THCTensor *dst) {
  if(self != dst)
    THC_copyTensor<real, real>(state, dst, self);

  THCTensor_free(state, self);
}

template <>
void THCTensor_copyIgnoringOverlaps<real>(THCState* state, THCTensor* dst, THCTensor* src) {
  // Called when we are copying into an overlapping index `dst`, but
  // we don't care which writer wins. Hacky but it works.
  // This is itself invoked by pointwiseApply2 / THCTensor_copy in
  // case that there are write overlaps.
  // FIXME: really, overlapping writes should be illegal/an error in Torch
  THC_pointwiseApply2<real, real>(
    state, dst, src,
    CopyOp<real, real>(),
    ReadOnly, /* ignore overwrites */
    ReadOnly);
}

THC_API void
THCTensor_(copyIgnoringOverlaps)(THCState* state, THCTensor* dst, THCTensor* src) {
  THCTensor_copyIgnoringOverlaps<real>(state, dst, src);
}

#define IMPLEMENT_THC_CUDA_TENSOR_COPY(TYPEC, TYPECUDA, SCALARC)        \
  THC_API void                                                          \
  THCTensor_(copyCuda##TYPEC)(THCState *state,                          \
                              THCTensor *self,                          \
                              THCuda##TYPECUDA##Tensor *src) {          \
    THC_copyTensor<real, SCALARC>(state, self, src); \
  }

IMPLEMENT_THC_CUDA_TENSOR_COPY(Byte, Byte, uint8_t)
IMPLEMENT_THC_CUDA_TENSOR_COPY(Char, Char, int8_t)
IMPLEMENT_THC_CUDA_TENSOR_COPY(Short, Short, int16_t)
IMPLEMENT_THC_CUDA_TENSOR_COPY(Int, Int, int32_t)
IMPLEMENT_THC_CUDA_TENSOR_COPY(Long, Long, int64_t)
// THCudaTensor aka the non-existent THCudaFloatTensor
IMPLEMENT_THC_CUDA_TENSOR_COPY(Float, , float)
IMPLEMENT_THC_CUDA_TENSOR_COPY(Double, Double, double)
#ifdef CUDA_HALF_TENSOR
IMPLEMENT_THC_CUDA_TENSOR_COPY(Half, Half, half)
#endif

#undef IMPLEMENT_THC_CUDA_TENSOR_COPY

#endif
