
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/ELU.cu"
#else

#include "../common.h"


void THNN_(ELU_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           accreal alpha,
           accreal scale,
           accreal input_scale,
           bool inplace)
{
  real negcoef = ScalarConvert<accreal, real>::to(alpha * scale);
  real poscoef = ScalarConvert<accreal, real>::to(scale * input_scale);
  real negiptcoef = ScalarConvert<accreal, real>::to(input_scale);
  THCUNN_assertSameGPU(state, 2, input, output);

  if (inplace)
  {
    THC_pointwiseApply1<real>(state, input, ELUupdateOutputIP_functor<real>(negcoef, poscoef, negiptcoef));
    THCTensor_(set)(state, output, input);
  }
  else
  {
    THCTensor_(resizeAs)(state, output, input);
    THC_pointwiseApply2<real, real>(state, output, input, ELUupdateOutput_functor<real>(negcoef, poscoef, negiptcoef));
  }
}


void THNN_(ELU_updateGradInput)(
           THCState *state,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *output,
           accreal alpha,
           accreal scale,
           accreal input_scale)
{
  real negcoef = ScalarConvert<accreal, real>::to(alpha * scale);
  real poscoef = ScalarConvert<accreal, real>::to(scale * input_scale);
  real negiptcoef = ScalarConvert<accreal, real>::to(input_scale);
  THCUNN_check_nElement(state, output, gradOutput);
  THCUNN_assertSameGPU(state, 3, output, gradOutput, gradInput);

  THCTensor_(resizeAs)(state, gradInput, output);
  THC_pointwiseApply3<real, real, real>(state, gradInput, output, gradOutput, ELUupdateGradInput_functor<real>(negcoef, poscoef, negiptcoef));
}

#endif
