#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <THC/THCNumerics.cuh>

template <typename T, typename U>
__global__ void half_test_kernel(const T* input, U* output) {
  if (input[0] < input[1] || input[0] >= input[1]) {
    output[0] = 123;
  }
}

at::Tensor half_test(at::Tensor input) {
  auto output = at::empty(1, input.options().dtype(at::kFloat));
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.type(), "half_test", [&] {
    half_test_kernel<scalar_t>
        <<<1, 1>>>(input.data<scalar_t>(), output.data<float>());
  });
  return output;
}
